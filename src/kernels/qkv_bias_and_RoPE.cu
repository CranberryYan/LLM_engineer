#include "hip/hip_runtime.h"
// linear: 用于gemm, 2d x 2d: fused
//          batch_gemm, 4d x 4d: [batch_size, head_num, seq_len, head_size]: Qk_gemm, Qk*v_gemm
//          LMHead(linear): hidden_units -> vocab_size(接下来要sampling)
// input: [num_tokens] -> input_embedding: [num_tokens, hidden_units](num_tokens: bs * q_len, q_len: 单个句子中的token集合, bs: 句子)
//                              |
//                              -> cal_paddingoffset: [bs, max_q_len, hidden_units]
//                              |
//                              -> build_casual_mask: mask: [bs, max_q_len, max_k_len]
//                              |
//                              -> RMSNorm: [num_tokens, hidden_units] -> fusedQkvGemm: * [hidden_units, hidden_units] -> [num_tokens, hidden_units]
//                              -> AddbiasAndPaddingAndRope: [max_num_tokens, hidden_units] -> [bs, q_head_num, max_q_len, head_size]  ->
//                                                                                          |
//                                                                                          -> [bs, kv_head_num, max_q_len, head_size] ->
//                                                                                          |
//                                                                                          -> [bs, kv_head_num, max_q_len, head_size] ->

// 注: Llama2中的RoPE, 是前一半与后一半为一组, 因此, 不可以向量化计算
// 1. add qkv_bias to QKV, which has shape[num tokens, qkv head num, head_size], k head num = v head num
// 2. padding, QKV splits to q k v and their shape is [bs, q head num(kv head num), max q len, head_size]
// 3. rope and do attention
// 4. write back to global memory
// 在多头注意力中, 模型的总维度(即 hidden_units)被分成多个独立的子空间, 每个子空间对应一个注意力头。
//  每个注意力头只会处理输入的一个子空间(即一个较小的维度), 因此每个头的维度为 head_size
// input: [num_tokens, hidden_units]
//      int token_id = blockIdx.x;
//      int head_id  = blockIdx.y;
//      int token_padding_offset = padding_offset[token_id];
//      int dst_token_id = token_id + token_padding_offset;
//      int batch_id = dst_token_id / seq_len; // 第几个seq_len
//  num_tokens -> bs * max_q_len
//  hidden_units -> q_head_num * head_size
// input:  QKV: [num_tokens, qkv_head_num, head_size]
//         qkv bias: [qkv_head_num, head_size]
// output: q: [bs, q_head_num, max_q_len, head_size]
//         k: [bs, kv_head_num, max_q_len, head_size]
//         v: [bs, kv_head_num, max_q_len, head_size]
// repeat kv
#include <math.h>
#include <stdio.h>
#include "src/kernels/qkv_bias_and_RoPE.h"

// hugging face
//    def _compute_inv_freq(self, base: Union[int, float]) -> torch.Tensor:
//         """Compute the inverse frequency."""
//         inv_freq = 1.0 / (base**(torch.arange(
//             0, self.rotary_dim, 2, dtype=torch.float, device="cuda") /
//                                  self.rotary_dim))
//         return inv_freq

//     def _compute_cos_sin_cache(self) -> torch.Tensor:
//         """Compute the cos and sin cache."""
//         inv_freq = self._compute_inv_freq(self.base)
//         t = torch.arange(self.max_position_embeddings,
//                          dtype=torch.float,
//                          device="cuda")

//         freqs = torch.einsum("i,j -> ij", t, inv_freq) // 外积
//         cos = freqs.cos() // 2048, 64
//         sin = freqs.sin()
//         cache = torch.cat((cos, sin), dim=-1)
//         return cache

// cos与sin中的角度(seita)
inline __device__ float2 GetRoPEfreq(int zid, int rot_embed_dim, float base, float t_step) {
    // 对应 HF 的 inv_freq
    float inv_freq = t_step / powf(base, zid / (float)rot_embed_dim);

    return {cos(inv_freq), sin(inv_freq)};
}

// 无向量化
inline __device__ float2 GetRoPEres(float data, float data_rotate, const float2 coef) {
    float2 rot_v; // rotary value
    rot_v.x = coef.x * data - coef.y * data_rotate;
    rot_v.y = coef.x * data_rotate + coef.y * data;

    return rot_v;
}

// this kernel for context attention
template <typename T>
__global__ void add_fusedQKV_bias_transpose_kernel(
    T *q_buf, T *k_buf, T *v_buf, T *QKV,
    const T *qkv_bias,
    const int *padding_offset,      // created before qkv linear
    const int *history_length,
    const int *input_length,        // actual length of each seq
    const int batch_size,
    const int seq_len,              // max_seq_len to pad to
    const int token_num,
    const int head_num,
    const int kv_head_num,
    const int head_size,
    const int rotary_embedding_dim,
    float rotary_embedding_base,    // default 10000 in llama
    int max_position_embeddings,    /*default 2048 in llama*/
    bool use_dynamic_ntk            /*placeholder for ntk RoPE*/) {
    int vec_size = Vec<T>::size;
    using Vec_t = typename Vec<T>::Type;
    // offset
    int token_id = blockIdx.x;
    int head_id  = blockIdx.y;
    int tid = threadIdx.x;
    int token_padding_offset = padding_offset[token_id];
    int dst_token_id = token_id + token_padding_offset;
    int batch_id = dst_token_id / seq_len;          // 第几个seq_len
    int local_token_id = dst_token_id % seq_len;    // seq_len维度层面, 具体那一个token
    int qkv_head_num = head_num + 2 * kv_head_num;  // q + k + v(k = v) 为什么 q != k or v

    //         blockIdx.x   blockIdx.y  threadIdx.x
    // input: [num tokens, qkv head num, head size]
    // 计算input -> 因此不使用padding_offset后的id
    int q_id = token_id * qkv_head_num * head_size + head_id * head_size + tid * vec_size;
    int k_id = token_id * qkv_head_num * head_size + head_id * head_size + tid * vec_size + head_num * head_size;

    // RoPE
    const int cur_seq_history_len = history_length[batch_id];
    const int time_step = cur_seq_history_len + local_token_id;
    if (tid >= rotary_embedding_dim / 2) {
        return; // 2 2一组的RoPE, 只需要前64个thread
    }
    float2 cos_sin = GetRoPEfreq(tid * 2, rotary_embedding_dim, rotary_embedding_base, time_step);
    float2 q_rotate = GetRoPEres(QKV[q_id], QKV[q_id + head_size / 2], cos_sin); // 返回 x[0] 和 x[64]
    float2 k_rotate = GetRoPEres(QKV[k_id], QKV[k_id + head_size / 2], cos_sin); // 返回 x[0] 和 x[64]

    // wrtie back
    // output: q: [bs, q_head_num, max_q_len, head_size]
    //         k: [bs, kv_head_num, max_q_len, head_size]
    //         v: [bs, kv_head_num, max_q_len, head_size]
    int dst_q_id  = batch_id * head_num * seq_len * head_size +
                    head_id * seq_len * head_size + 
                    local_token_id * head_size + tid;
    int dst_kv_id = batch_id * kv_head_num * seq_len * head_size +
                    head_id * seq_len * head_size + 
                    local_token_id * head_size + tid;

    q_buf[dst_q_id] = q_rotate.x;
    q_buf[dst_q_id + head_size / 2] = q_rotate.y;
    if (head_id < kv_head_num) {
        // for MQA and GQA
        k_buf[dst_kv_id] = k_rotate.x;
        k_buf[dst_kv_id + head_size / 2] = k_rotate.y;
    }
}

// // this kernel for masked self  attention
template <typename T>
__global__ void rope_kernel_for_self_decoder(T *q, T *k, const int batch_szie, 
    const int head_num, const int kv_head_num, const int head_size, const int step, 
    int rotary_embedding_dim, float rotary_embedding_base) {

    int tid = threadIdx.x;
    int q_head_id = blockIdx.x;
    int q_batch_id = blockIdx.y;
    int kv_head_id = q_head_id / (head_num / kv_head_num);
    int kv_batch_id = q_batch_id;
    int batch_stride = head_num * head_size;
    int kv_batch_stride = kv_head_num * head_size;
    int head_sride = head_size;
    int q_offset = q_batch_id * batch_stride + q_head_id * head_sride + tid;
    int k_offset = kv_batch_id * batch_stride + kv_head_id * head_sride + tid;

    if (tid >= rotary_embedding_dim / 2) {
        return;
    }
    float2 cos_sin = GetRoPEfreq(tid * 2, rotary_embedding_dim, rotary_embedding_base, step - 1);
    float2 q_rotate = GetRoPEres(q[q_offset], q[q_offset + head_size / 2], cos_sin); // 返回 x[0] 和 x[64]
    float2 k_rotate = GetRoPEres(k[k_offset], k[k_offset + head_size / 2], cos_sin); // 返回 x[0] 和 x[64]

    q[q_offset] = q_rotate.x;
    q[q_offset + head_size / 2] = q_rotate.y;
    k[q_offset] = k_rotate.x;
    k[k_offset + head_size / 2] = k_rotate.y;
    }

// TODO
template <>
__global__ void rope_kernel_for_self_decoder(half *q, half*k, 
    const int batch_szie, const int head_num, const int kv_head_num, const int head_size, const int step, 
    int rotary_embedding_dim, float rotary_embedding_base) {

    }

// input: [num_tokens, hidden_units]
// output: q: [bs, head_num, max_q_len, head size]
//       k/v: [bs, kv head_num, max_q_len, head size]
// this launch for context attention
template <typename T>
void launchAddFusedQKVBiasTransposeAndRoPE(
    TensorWrapper<T> *q_buf, TensorWrapper<T> *k_buf, TensorWrapper<T> *v_buf,
    TensorWrapper<T> *QKV, BaseWeight<T> &qkv,
    TensorWrapper<int> *padding_offset, TensorWrapper<int> *history_length, TensorWrapper<int> *input_length,
    LLaMaAttentionStaticParams &params) {
    int token_num = QKV->shape[0];
    int qkv_head_num = QKV->shape[1];
    int head_size = QKV->shape[2];
    int batch_size = q_buf->shape[0];
    int head_num = q_buf->shape[1];
    int seq_len = q_buf->shape[2];
    int kv_head_num = (qkv_head_num - head_num) / 2;

    // block只需要考虑head size, 来进行维度的分配
    dim3 grid(token_num, head_num);
    dim3 block(head_size);

    add_fusedQKV_bias_transpose_kernel<T><<<grid, block>>>(
            q_buf->data, k_buf->data, v_buf->data, QKV->data,
            qkv.bias, padding_offset->data, history_length->data, input_length->data,
            batch_size, seq_len, token_num, head_num, kv_head_num, head_size,
            params.rotary_embedding_dim, params.rotary_embedding_base, 
            params.max_position_embeddings, params.use_dynamic_ntk);
}

// this launch for masked self  attention
template<typename T>
void launchRoPE(TensorWrapper<T> *qkv_buf, TensorWrapper<int> *step, 
    LLaMaAttentionStaticParams &static_params) {
    const int batch_size = qkv_buf->shape[0];
    const int qkv_head_num = qkv_buf->shape[1];
    const int head_num = 32; // only for llama2
    const int head_size = qkv_buf->shape[2];
    const int cur_step = step->getVal();
    LLM_CHECK(batch_size == 1);
    LLM_CHECK(qkv_head_num == 96);
    LLM_CHECK(head_size == 128);
    T *qkv_data = qkv_buf->data;
    T *q = qkv_data;
    T *k = qkv_data + head_num * head_size;

    int rotary_embedding_dim = static_params.rotary_embedding_dim;
    float rotary_embedding_base = static_params.rotary_embedding_base;
    int max_position_embeddings = static_params.max_position_embeddings;
    dim3 grid(head_num, batch_size);
    dim3 block(head_size);

    // MHA: q_head_num == kv_head_num
    rope_kernel_for_self_decoder<T><<<grid, block>>>(
        q, k, batch_size, head_num, head_num, head_size,
        cur_step, rotary_embedding_dim, rotary_embedding_base);
}

template void launchAddFusedQKVBiasTransposeAndRoPE(
    TensorWrapper<float> *q_buf, TensorWrapper<float> *k_buf, TensorWrapper<float> *v_buf,
    TensorWrapper<float> *QKV, BaseWeight<float> &qkv,
    TensorWrapper<int> *padding_offset, TensorWrapper<int> *history_length, TensorWrapper<int> *input_length,
    LLaMaAttentionStaticParams &params);
// template void launchAddFusedQKVBiasTransposeAndRoPE(
//             TensorWrapper<half> *q_buf, TensorWrapper<half> *k_buf, TensorWrapper<half> *v_buf,
//             TensorWrapper<half> *QKV, BaseWeight<half> &qkv,
//             TensorWrapper<int> *padding_offset, TensorWrapper<int> *history_length, TensorWrapper<int> *input_length,
//             LLaMaAttentionStaticParams &params);

template void launchRoPE(
    TensorWrapper<float>* qkv_buf, TensorWrapper<int>* step,
    LLaMaAttentionStaticParams& static_params);